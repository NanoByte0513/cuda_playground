#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cutlass/half.h>
#include <cutlass/cutlass.h>
#include <cutlass/aligned_buffer.h>
#include <cute/tensor.hpp>
#include <cutlass/gemm/warp/default_mma_tensor_op_sm80.h>
#include <cutlass/util/host_tensor.h>
#include "hip/hip_fp16.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "utils/utils.cuh"
#include <fcntl.h>
#include <unistd.h>
#include <random>

#define LEN_M 16
#define LEN_N 8
#define LEN_K 32

using ElementAccumulator = float;
using ElementComputeEpilogue = ElementAccumulator;
using ElementInputA = cutlass::half_t;
using ElementInputB = cutlass::half_t;
using ElementOutput = float;

using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::ColumnMajor; // 这里的列主序并不是指B的数据在内存中是按列主序排列的，而是B要按照列主序访问
using LayoutOutput = cutlass::layout::RowMajor;

using ThreadblockShape = cutlass::gemm::GemmShape<LEN_M, LEN_N, LEN_K>; // 这里指的是一个block沿K维度滑动，每一次要从AB读取的数据大小(From gmem to smem)
using WarpShape = cutlass::gemm::GemmShape<16, 8, 16>; // 这里指的是一个warp每次从smem读取的数据大小(From smem to reg)
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>; // 这里指一次TensorCore指令读取的数据大小
using Mma = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      WarpShape, InstructionShape, ElementInputA, LayoutInputA, ElementInputB, LayoutInputB, ElementOutput,
      cutlass::layout::RowMajor>::Type;


using float16 = uint16_t;
/// Test kernel
__global__ void kernel(
    typename Mma::ElementC *output_C, 
    typename Mma::ElementA const *input_A,
    typename Mma::ElementB const *input_B) {

    

    // Use AlignedBuffer to store trivially copyable objects in unions and __shared__ buffers.
    __shared__ cutlass::AlignedBuffer<
        typename Mma::ElementA, ThreadblockShape::kM * ThreadblockShape::kK> smem_buffer_A;

    __shared__ cutlass::AlignedBuffer<
        typename Mma::ElementB, ThreadblockShape::kN * ThreadblockShape::kK> smem_buffer_B;

    // Read whole block to smem
    if (threadIdx.x == 0) {
        typename Mma::ElementA *smem_ptr_A = smem_buffer_A.data();
        #pragma unroll 1
        for (size_t i = 0; i < smem_buffer_A.size(); ++i) {
            cutlass::ReferenceFactory<typename Mma::ElementA>::get(smem_ptr_A, i) =
                cutlass::ReferenceFactory<typename cutlass::platform::remove_const<
                    typename Mma::ElementA>::type>::get(input_A, i);
        }

        typename Mma::ElementB *smem_ptr_B = smem_buffer_B.data();
        #pragma unroll 1
        for (size_t i = 0; i < smem_buffer_B.size(); ++i) {
            cutlass::ReferenceFactory<typename Mma::ElementB>::get(smem_ptr_B, i) =
                cutlass::ReferenceFactory<typename cutlass::platform::remove_const<
                    typename Mma::ElementB>::type>::get(input_B, i);
        }
    }

    __syncthreads();

    // float tempA[LEN_M * LEN_K];
    // cutlass::half_t* smemA_h = (cutlass::half_t*)(smem_buffer_A.raw_data());
    // for (size_t i = 0; i < LEN_M * LEN_K; ++i) {
    //     tempA[i] = static_cast<float>(smemA_h[i]);
    // }

    // float tempB[LEN_K * LEN_N];
    // cutlass::half_t* smemB_h = (cutlass::half_t*)(smem_buffer_B.raw_data());
    // for (size_t i = 0; i < LEN_K * LEN_N; ++i) {
    //     tempB[i] = static_cast<float>(smemB_h[i]);
    // }

    //
    // Construct warp-level matrix product
    //

    using FragmentA = typename Mma::FragmentA;
    using FragmentB = typename Mma::FragmentB;
    using FragmentC = typename Mma::FragmentC;

    typename Mma::LayoutA layout_A = Mma::LayoutA::packed({ThreadblockShape::kM, ThreadblockShape::kK});
    typename Mma::LayoutB layout_B = Mma::LayoutB::packed({ThreadblockShape::kK, ThreadblockShape::kN});
    typename Mma::LayoutC layout_C = Mma::LayoutC::packed({Mma::Shape::kM, Mma::Shape::kN}); // Mma::Shape实际上是WarpShape而不是InstructionShape

    typename Mma::IteratorA iter_A({smem_buffer_A.data(), layout_A}, cutlass::arch::LaneId());
    typename Mma::IteratorB iter_B({smem_buffer_B.data(), layout_B}, cutlass::arch::LaneId());

    FragmentA frag_A;
    FragmentB frag_B;
    FragmentC accum;
    accum.clear();

    Mma mma;
    CUTLASS_PRAGMA_UNROLL
    for (int k = 0; k < ThreadblockShape::kK; k += Mma::Policy::MmaShape::kK) { // Mma::Policy::MmaShape是InstructionShape
        iter_A.load(frag_A);
        iter_B.load(frag_B);

        ++iter_A;
        ++iter_B;

        mma(accum, frag_A, frag_B, accum);
    }
  
    typename Mma::IteratorC iter_C({output_C, layout_C}, cutlass::arch::LaneId());

    iter_C.store(accum);
}



int main() {
    // Create a tuple of problem size for matrix multiplication
    cutlass::gemm::GemmCoord problem_size(LEN_M, LEN_N, LEN_K);

    // Initialize tensors using CUTLASS helper functions
    cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(
        problem_size.mk());  // <- Create matrix A with dimensions M x K
    cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(
        problem_size.kn());  // <- Create matrix B with dimensions K x N
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(
        problem_size.mn());
    cutlass::reference::host::TensorFill(
        tensor_d.host_view());  // <- fill matrix D on host with zeros


    
    cutlass::half_t *h_A = new cutlass::half_t[LEN_M * LEN_K];
    cutlass::half_t *h_B = new cutlass::half_t[LEN_K * LEN_N];
    cutlass::half_t *h_C = new cutlass::half_t[LEN_M * LEN_N];  // cuBLAS计算结果

    // 初始化输入矩阵（随机值）
    for (int i = 0; i < LEN_M * LEN_K; i++) h_A[i] = static_cast<cutlass::half_t>(static_cast<float>(rand()) / RAND_MAX);
    for (int i = 0; i < LEN_K * LEN_N; i++) h_B[i] = static_cast<cutlass::half_t>(static_cast<float>(rand()) / RAND_MAX);

    // 设备内存分配（GPU）
    cutlass::half_t *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, LEN_M * LEN_K * sizeof(cutlass::half_t));
    hipMalloc((void**)&d_B, LEN_K * LEN_N * sizeof(cutlass::half_t));
    hipMalloc((void**)&d_C, LEN_M * LEN_N * sizeof(cutlass::half_t));

    // 数据从主机复制到设备
    hipMemcpy(d_A, h_A, LEN_M * LEN_K * sizeof(cutlass::half_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, LEN_K * LEN_N * sizeof(cutlass::half_t), hipMemcpyHostToDevice);

    // 复制数据到 tensor_a
    size_t num_elements = tensor_a.size();
    size_t size_in_bytes = num_elements * sizeof(ElementInputA);
    std::memcpy(tensor_a.host_data(), h_A, size_in_bytes);

    // 复制数据到 tensor_b
    num_elements = tensor_b.size();
    size_in_bytes = num_elements * sizeof(ElementInputB);
    std::memcpy(tensor_b.host_data(), h_B, size_in_bytes);

    // Copy data from host to GPU
    tensor_a.sync_device();
    tensor_b.sync_device();
    tensor_d.sync_device();

    kernel<<<dim3(1, 1, 1), dim3(32, 1, 1)>>>(tensor_d.device_data(), tensor_a.device_data(), tensor_b.device_data());
    hipDeviceSynchronize();
    tensor_d.sync_host();


    // 创建cuBLAS句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    // 定义标量参数（alpha=1, beta=0: C = A*B）
    float alpha = 1.0f;
    float beta = 0.0f;

    // 调用cuBLAS半精度矩阵乘法（hipblasGemmEx
    hipblasGemmEx(
        handle,
        HIPBLAS_OP_T,                     // A不转置（行优先）
        HIPBLAS_OP_N,                     // B不转置（行优先）
        LEN_M, LEN_N, LEN_K,             
        &alpha,                           // alpha标量
        d_A, HIP_R_16F, LEN_K,              // A矩阵（行优先，列数=K）
        d_B, HIP_R_16F, LEN_K,              // B矩阵（行优先，列数=N）
        &beta,                            // beta标量
        d_C, HIP_R_16F, LEN_M,              // 输出C（行优先，列数=N）
        HIP_R_32F,                      // 内部计算精度（float避免累积误差）
        CUBLAS_GEMM_DEFAULT_TENSOR_OP    // 使用Tensor Core
    );
    // 这里得到的C矩阵结果是对的，但是是转置后的（按列存储的）

    hipMemcpy(h_C, d_C, LEN_M * LEN_N * sizeof(cutlass::half_t), hipMemcpyDeviceToHost);

    bool noError = true;
    for(int i = 0; i < LEN_M * LEN_N; ++i) {
        float cub_val = h_C[(i % LEN_N) * LEN_M + i / LEN_N];
        float ker_val = tensor_d.host_data()[i];
        float diff = fabs(cub_val - ker_val);
        if(diff > 1e-2) {
            printf("[%d]: cub_val = %.4f, ker_val = %.4f, diff = %.4f\n", i, cub_val, ker_val, diff);
            noError = false;
        }
    }
    if(noError)
        printf("no error\n");



    // 释放资源
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    return 0;
}