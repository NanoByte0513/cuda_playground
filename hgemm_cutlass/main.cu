#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cutlass/half.h>
#include <cutlass/cutlass.h>
#include <cutlass/aligned_buffer.h>
#include <cute/tensor.hpp>
#include <cutlass/gemm/warp/default_mma_tensor_op_sm80.h>
#include <cutlass/util/host_tensor.h>
#include "hip/hip_fp16.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "utils/utils.cuh"
#include <fcntl.h>
#include <unistd.h>
#include <random>

#define LEN_M 16
#define LEN_N 8
#define LEN_K 512

using ElementAccumulator = float;
using ElementComputeEpilogue = ElementAccumulator;
using ElementInputA = cutlass::half_t;
using ElementInputB = cutlass::half_t;
using ElementOutput = float;

using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::ColumnMajor; // 这里的列主序并不是指B的数据在内存中是按列主序排列的，而是B要按照列主序访问
using LayoutOutput = cutlass::layout::RowMajor;

using ThreadblockShape = cutlass::gemm::GemmShape<LEN_M, LEN_N, LEN_K>; // 这里指的是一个block沿K维度滑动，每一次要从AB读取的数据大小(From gmem to smem)
using WarpShape = cutlass::gemm::GemmShape<16, 8, 16>; // 这里指的是一个warp每次从smem读取的数据大小(From smem to reg)
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>; // 这里指一次TensorCore指令读取的数据大小
using Mma = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      WarpShape, InstructionShape, ElementInputA, LayoutInputA, ElementInputB, LayoutInputB, ElementOutput,
      cutlass::layout::RowMajor>::Type;


using float16 = uint16_t;
/// Test kernel
__global__ void kernel(
    typename Mma::ElementC *output_C, 
    typename Mma::ElementA const *input_A,
    typename Mma::ElementB const *input_B) {
    cute::Tensor gmem_tensorA = cute::make_tensor(cute::make_gmem_ptr(input_A), cute::make_shape(ThreadblockShape::kM, ThreadblockShape::kK), cute::LayoutRight{});
    cute::Tensor gmem_tensorB = cute::make_tensor(cute::make_gmem_ptr(input_B), cute::make_shape(ThreadblockShape::kK, ThreadblockShape::kN), cute::LayoutRight{});
    cute::Tensor gmem_tensorC = cute::make_tensor(cute::make_gmem_ptr(output_C), cute::make_shape(ThreadblockShape::kM, ThreadblockShape::kN), cute::LayoutRight{});

    // Use AlignedBuffer to store trivially copyable objects in unions and __shared__ buffers.
    __shared__ cutlass::AlignedBuffer<
        typename Mma::ElementA, ThreadblockShape::kM * ThreadblockShape::kK> smem_buffer_A;

    __shared__ cutlass::AlignedBuffer<
        typename Mma::ElementB, ThreadblockShape::kN * ThreadblockShape::kK> smem_buffer_B;
    
    // Use the smem_buffer to init cute::tensors
    cute::Tensor smem_tensorA = cute::make_tensor(cute::make_smem_ptr(smem_buffer_A.data()), cute::make_shape(ThreadblockShape::kM, ThreadblockShape::kK), cute::LayoutRight{});
    cute::Tensor smem_tensorB = cute::make_tensor(cute::make_smem_ptr(smem_buffer_B.data()), cute::make_shape(ThreadblockShape::kK, ThreadblockShape::kN), cute::LayoutRight{});

    // Read whole block to smem
    cute::copy(gmem_tensorA, smem_tensorA);
    cute::copy(gmem_tensorB, smem_tensorB);

    __syncthreads();

    //
    // Construct warp-level matrix product
    //

    using FragmentA = typename Mma::FragmentA;
    using FragmentB = typename Mma::FragmentB;
    using FragmentC = typename Mma::FragmentC;

    typename Mma::LayoutA layout_A = Mma::LayoutA::packed({ThreadblockShape::kM, ThreadblockShape::kK});
    typename Mma::LayoutB layout_B = Mma::LayoutB::packed({ThreadblockShape::kK, ThreadblockShape::kN});
    typename Mma::LayoutC layout_C = Mma::LayoutC::packed({Mma::Shape::kM, Mma::Shape::kN}); // Mma::Shape实际上是WarpShape而不是InstructionShape

    typename Mma::IteratorA iter_A({smem_buffer_A.data(), layout_A}, cutlass::arch::LaneId());
    typename Mma::IteratorB iter_B({smem_buffer_B.data(), layout_B}, cutlass::arch::LaneId());

    FragmentA frag_A;
    FragmentB frag_B;
    FragmentC accum;
    accum.clear();

    Mma mma;
    CUTLASS_PRAGMA_UNROLL
    for (int k = 0; k < ThreadblockShape::kK; k += Mma::Policy::MmaShape::kK) { // Mma::Policy::MmaShape is InstructionShape
        iter_A.load(frag_A);
        iter_B.load(frag_B);

        ++iter_A;
        ++iter_B;

        mma(accum, frag_A, frag_B, accum);
    }
  
    typename Mma::IteratorC iter_C({output_C, layout_C}, cutlass::arch::LaneId());

    iter_C.store(accum);
}



int main() {
    // Create a tuple of problem size for matrix multiplication
    cutlass::gemm::GemmCoord problem_size(LEN_M, LEN_N, LEN_K);

    // Initialize tensors using CUTLASS helper functions
    cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(
        problem_size.mk());  // <- Create matrix A with dimensions M x K
    cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(
        problem_size.kn());  // <- Create matrix B with dimensions K x N
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(
        problem_size.mn());
    cutlass::reference::host::TensorFill(
        tensor_d.host_view());  // <- fill matrix D on host with zeros


    
    cutlass::half_t *h_A = new cutlass::half_t[LEN_M * LEN_K];
    cutlass::half_t *h_B = new cutlass::half_t[LEN_K * LEN_N];
    cutlass::half_t *h_C = new cutlass::half_t[LEN_M * LEN_N];  // cuBLAS计算结果

    // 初始化输入矩阵（随机值）
    for (int i = 0; i < LEN_M * LEN_K; i++) h_A[i] = static_cast<cutlass::half_t>(static_cast<float>(rand() - RAND_MAX / 2) / RAND_MAX * 2);
    for (int i = 0; i < LEN_K * LEN_N; i++) h_B[i] = static_cast<cutlass::half_t>(static_cast<float>(rand() - RAND_MAX / 2) / RAND_MAX * 2);

    // 设备内存分配（GPU）
    cutlass::half_t *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, LEN_M * LEN_K * sizeof(cutlass::half_t));
    hipMalloc((void**)&d_B, LEN_K * LEN_N * sizeof(cutlass::half_t));
    hipMalloc((void**)&d_C, LEN_M * LEN_N * sizeof(cutlass::half_t));

    // 数据从主机复制到设备
    hipMemcpy(d_A, h_A, LEN_M * LEN_K * sizeof(cutlass::half_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, LEN_K * LEN_N * sizeof(cutlass::half_t), hipMemcpyHostToDevice);

    // 复制数据到 tensor_a
    size_t num_elements = tensor_a.size();
    size_t size_in_bytes = num_elements * sizeof(ElementInputA);
    std::memcpy(tensor_a.host_data(), h_A, size_in_bytes);

    // 复制数据到 tensor_b
    num_elements = tensor_b.size();
    size_in_bytes = num_elements * sizeof(ElementInputB);
    std::memcpy(tensor_b.host_data(), h_B, size_in_bytes);

    // Copy data from host to GPU
    tensor_a.sync_device();
    tensor_b.sync_device();
    tensor_d.sync_device();

    kernel<<<dim3(1, 1, 1), dim3(32, 1, 1)>>>(tensor_d.device_data(), tensor_a.device_data(), tensor_b.device_data());
    hipDeviceSynchronize();
    tensor_d.sync_host();


    // 创建cuBLAS句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    // 定义标量参数（alpha=1, beta=0: C = A*B）
    float alpha = 1.0f;
    float beta = 0.0f;

    // 调用cuBLAS半精度矩阵乘法（hipblasGemmEx
    hipblasGemmEx(
        handle,
        HIPBLAS_OP_T,                     // A要转置，因为cublass默认列主序
        HIPBLAS_OP_N,                     // B不转置，保持
        LEN_M, LEN_N, LEN_K,             
        &alpha,                           
        d_A, HIP_R_16F, LEN_K,              // 列数K
        d_B, HIP_R_16F, LEN_K,              // 
        &beta,                            
        d_C, HIP_R_16F, LEN_M,              // 行数M
        HIP_R_32F,                      // 内部计算精度（float避免累积误差）
        CUBLAS_GEMM_DEFAULT_TENSOR_OP    // 使用Tensor Core
    );
    // 这里得到的C矩阵结果是对的，但是是转置后的（按列存储的）

    hipMemcpy(h_C, d_C, LEN_M * LEN_N * sizeof(cutlass::half_t), hipMemcpyDeviceToHost);

    bool noError = true;
    for(int i = 0; i < LEN_M * LEN_N; ++i) {
        float cub_val = h_C[(i % LEN_N) * LEN_M + i / LEN_N];
        float ker_val = tensor_d.host_data()[i];
        float diff = fabs(cub_val - ker_val);
        if(diff > 1e-2) {
            printf("[%d]: cub_val = %.4f, ker_val = %.4f, diff = %.4f\n", i, cub_val, ker_val, diff);
            noError = false;
        }
    }
    if(noError)
        printf("no error\n");



    // 释放资源
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    return 0;
}